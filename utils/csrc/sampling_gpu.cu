#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "sampling_gpu.h"
#include "cuda_utils.h"

// input: points(b, n, c) idx(b, m)
// output: out(b, m, c)
__global__ void gather_points_kernel(int b, int n, int c, int m,
				     const float *__restrict__ points,
				     const int *__restrict__ idx,
				     float *__restrict__ out) {
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
	for (int j = blockIdx.y * blockDim.x + threadIdx.x; j < m;
	     j += blockDim.x * gridDim.y) {
	    int a = idx[i * m + j];
	    memcpy(out + (i * m + j) * c, points + (i * n + a) * c,
		   sizeof(float) * c);
	}
    }
}

void gather_points_kernel_wrapper(int b, int n, int c, int npoints,
				  const float *points, const int *idx,
				  float *out, hipStream_t stream) {

    hipError_t err;
    gather_points_kernel<<<dim3(2, 8, 1), opt_n_threads(npoints) / 4, 0,
			   stream>>>(b, n, c, npoints, points, idx, out);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
	exit(-1);
    }
}

__global__ void furthest_point_sampling_kernel(
    int b, int n, int m, const float *__restrict__ dataset,
    float *__restrict__ temp, int *__restrict__ idxs) {
    if (m <= 0)
	return;
    const int BlockSize = 512;
    __shared__ float dists[BlockSize];
    __shared__ int dists_i[BlockSize];
    const int BufferSize = 3072;
    __shared__ float buf[BufferSize * 3];

    for (int i = blockIdx.x; i < b; i += gridDim.x) {
	int old = 0;
	if (threadIdx.x == 0)
	    idxs[i * m + 0] = old;
	for (int j = threadIdx.x; j < n; j += blockDim.x) {
	    temp[blockIdx.x * n + j] = 1e38;
	}
	for (int j = threadIdx.x; j < min(BufferSize, n) * 3; j += blockDim.x) {
	    buf[j] = dataset[i * n * 3 + j];
	}
	__syncthreads();
	for (int j = 1; j < m; j++) {
	    int besti = 0;
	    float best = -1;
	    float x1 = dataset[i * n * 3 + old * 3 + 0];
	    float y1 = dataset[i * n * 3 + old * 3 + 1];
	    float z1 = dataset[i * n * 3 + old * 3 + 2];
	    for (int k = threadIdx.x; k < n; k += blockDim.x) {
		float td = temp[blockIdx.x * n + k];
		float x2, y2, z2;
		if (k < BufferSize) {
		    x2 = buf[k * 3 + 0];
		    y2 = buf[k * 3 + 1];
		    z2 = buf[k * 3 + 2];
		} else {
		    x2 = dataset[i * n * 3 + k * 3 + 0];
		    y2 = dataset[i * n * 3 + k * 3 + 1];
		    z2 = dataset[i * n * 3 + k * 3 + 2];
		}
		float d = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) +
			  (z2 - z1) * (z2 - z1);
		float d2 = min(d, td);
		if (d2 != td)
		    temp[blockIdx.x * n + k] = d2;
		if (d2 > best) {
		    best = d2;
		    besti = k;
		}
	    }
	    dists[threadIdx.x] = best;
	    dists_i[threadIdx.x] = besti;
	    for (int u = 0; (1 << u) < blockDim.x; u++) {
		__syncthreads();
		if (threadIdx.x < (blockDim.x >> (u + 1))) {
		    int i1 = (threadIdx.x * 2) << u;
		    int i2 = (threadIdx.x * 2 + 1) << u;
		    if (dists[i1] < dists[i2]) {
			dists[i1] = dists[i2];
			dists_i[i1] = dists_i[i2];
		    }
		}
	    }
	    __syncthreads();
	    old = dists_i[0];
	    if (threadIdx.x == 0)
		idxs[i * m + j] = old;
	}
    }
}

void furthest_point_sampling_kernel_wrapper(int b, int n, int m,
					    const float *dataset, float *temp,
					    int *idxs, hipStream_t stream) {

    hipError_t err;
    furthest_point_sampling_kernel<<<b, opt_n_threads(n), 0, stream>>>(
	b, n, m, dataset, temp, idxs);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
	exit(-1);
    }
}
